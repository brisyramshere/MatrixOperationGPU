#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include"hip/hip_runtime.h"
 
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int dim0, unsigned int dim1);
 
__global__ void addKernel(int *c, const int *a, const int *b)
{
	//int i =(blockIdx.z*blockDim.x*blockDim.y+blockIdx.y*blockDim.x + blockIdx.x)+threadIdx.x;
	int i = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x;
	c[i] = powf(2 * a[i] * b[i],2);
}
 
extern "C" double addnum(int *c, const int *a, const int *b, unsigned int dim0, unsigned int dim1)
{
	hipError_t cudaStatus = addWithCuda(c, a, b, dim0, dim1);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}
 
}
 
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int dim0,unsigned dim1)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	int size = dim0*dim1;
	hipError_t cudaStatus;
 
	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}
 
	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
 
	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
 
	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
 
	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
 
	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
 
	// Launch a kernel on the GPU with one thread for each element.
	dim3 DG(dim0 / 512, 512);
	addKernel<<<DG, 512>>>(dev_c, dev_a, dev_b);
 
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
 
	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}
 
	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
 
Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);
 
	return cudaStatus;
}